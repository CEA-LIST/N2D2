#include "hip/hip_runtime.h"
/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include <hip/hip_fp16.h>

#include "Activation/Activation_CUDA_Kernels.hpp"

// LeakyRectifier
__global__ void cudaHRectifier_propagate_kernel(__half* x,
                                                unsigned int size,
                                                __half leakSlope,
                                                int shifting,
                                                __half clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] = __float2half(__half2float(x[i]) / (1 << shifting));
        else if (shifting < 0)
            x[i] = __float2half(__half2float(x[i]) * (1 << (-shifting)));

        if (__half2float(clipping) > 0.0f) {
#if __CUDA_ARCH__ >= 530
            x[i] = (__half2float(x[i]) > 0.0f)
                ? ((__hlt(x[i], clipping))
                    ? x[i]
                    : clipping)
                : __hmul(leakSlope, x[i]);
#else
            x[i] = (__half2float(x[i]) > 0.0f)
                ? ((__half2float(x[i]) < __half2float(clipping))
                    ? x[i]
                    : clipping)
                : __float2half(__half2float(leakSlope) * __half2float(x[i]));
#endif
        }
        else
#if __CUDA_ARCH__ >= 530
            x[i] = (__half2float(x[i]) > 0.0f) ? x[i] : __hmul(leakSlope, x[i]);
#else
            x[i] = (__half2float(x[i]) > 0.0f) ? x[i]
                : __float2half(__half2float(leakSlope) * __half2float(x[i]));
#endif
    }
}

__global__ void cudaSRectifier_propagate_kernel(float* x,
                                                unsigned int size,
                                                float leakSlope,
                                                int shifting,
                                                float clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] /= (1 << shifting);
        else if (shifting < 0)
            x[i] *= (1 << (-shifting));

        if (clipping > 0.0f)
            x[i] = (x[i] > 0.0f) ? min(x[i], clipping) : leakSlope * x[i];
        else
            x[i] = (x[i] > 0.0f) ? x[i] : leakSlope * x[i];
    }
}

__global__ void cudaDRectifier_propagate_kernel(double* x,
                                                unsigned int size,
                                                double leakSlope,
                                                int shifting,
                                                double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] /= (1 << shifting);
        else if (shifting < 0)
            x[i] *= (1 << (-shifting));

        if (clipping > 0.0)
            x[i] = (x[i] > 0.0) ? min(x[i], clipping) : leakSlope * x[i];
        else
            x[i] = (x[i] > 0.0) ? x[i] : leakSlope * x[i];
    }
}

__global__ void cudaHRectifier_backPropagate_kernel(__half* x,
                                                    __half* dx,
                                                    unsigned int size,
                                                    __half leakSlope,
                                                    int shifting,
                                                    __half clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] = __float2half(__half2float(dx[i]) / (1 << shifting));
        else if (shifting < 0)
            dx[i] = __float2half(__half2float(dx[i]) * (1 << (-shifting)));

        if (__half2float(clipping) > 0.0f) {
#if __CUDA_ARCH__ >= 530
            dx[i] = (__hgt(x[i], clipping))
                ? __float2half(0.0f)
                : (__half2float(x[i]) > 0.0f)
                    ? dx[i]
                    : __hmul(leakSlope, dx[i]);
#else
            dx[i] = (__half2float(x[i]) > __half2float(clipping))
                ? __float2half(0.0f)
                : (__half2float(x[i]) > 0.0f)
                    ? dx[i]
                    : __float2half(__half2float(leakSlope)
                                   * __half2float(dx[i]));
#endif
        }
        else {
#if __CUDA_ARCH__ >= 530
            dx[i] = (__half2float(x[i]) > 0.0f) ? dx[i]
                                                : __hmul(leakSlope, dx[i]);
#else
            dx[i] = (__half2float(x[i]) > 0.0f) ? dx[i]
                : __float2half(__half2float(leakSlope) * __half2float(dx[i]));
#endif
        }
    }
}

__global__ void cudaSRectifier_backPropagate_kernel(float* x,
                                                    float* dx,
                                                    unsigned int size,
                                                    float leakSlope,
                                                    int shifting,
                                                    float clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] /= (1 << shifting);
        else if (shifting < 0)
            dx[i] *= (1 << (-shifting));

        if (clipping > 0.0) {
            dx[i] *= (x[i] > clipping) ? 0.0f : (x[i] > 0.0f)
                                       ? 1.0f
                                       : leakSlope;
        }
        else
            dx[i] *= (x[i] > 0.0f) ? 1.0f : leakSlope;
    }
}

__global__ void cudaDRectifier_backPropagate_kernel(double* x,
                                                    double* dx,
                                                    unsigned int size,
                                                    double leakSlope,
                                                    int shifting,
                                                    double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] /= (1 << shifting);
        else if (shifting < 0)
            dx[i] *= (1 << (-shifting));

        if (clipping > 0.0) {
            dx[i] *= (x[i] > clipping) ? 0.0 : (x[i] > 0.0)
                                       ? 1.0
                                       : leakSlope;
        }
        else
            dx[i] *= (x[i] > 0.0) ? 1.0 : leakSlope;
    }
}

// Saturation
__global__ void cudaHSaturation_propagate_kernel(__half* x,
                                                 unsigned int size,
                                                 int shifting,
                                                 __half threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] = __float2half(__half2float(x[i]) / (1 << shifting));
        else if (shifting < 0)
            x[i] = __float2half(__half2float(x[i]) * (1 << (-shifting)));

#if __CUDA_ARCH__ >= 530
        x[i] = (__hlt(x[i], __hneg(threshold))) ? __hneg(threshold)
             : (__hgt(x[i], threshold)) ? threshold
             : x[i];
#else
        x[i] = (__half2float(x[i]) < -__half2float(threshold))
             ? __float2half(-__half2float(threshold))
                : (__half2float(x[i]) > __half2float(threshold)) ? threshold
                : x[i];
#endif
    }
}

__global__ void cudaSSaturation_propagate_kernel(float* x,
                                                 unsigned int size,
                                                 int shifting,
                                                 float threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] /= (1 << shifting);
        else if (shifting < 0)
            x[i] *= (1 << (-shifting));

        x[i] = (x[i] < -threshold) ? -threshold
             : (x[i] > threshold) ? threshold
             : x[i];
    }
}

__global__ void cudaDSaturation_propagate_kernel(double* x,
                                                 unsigned int size,
                                                 int shifting,
                                                 double threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            x[i] /= (1 << shifting);
        else if (shifting < 0)
            x[i] *= (1 << (-shifting));

        x[i] = (x[i] < -threshold) ? -threshold
             : (x[i] > threshold) ? threshold
             : x[i];
    }
}

__global__ void
cudaHSaturation_backPropagate_kernel(__half* x,
                                     __half* dx,
                                     unsigned int size,
                                     int shifting,
                                     __half threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] = __float2half(__half2float(dx[i]) / (1 << shifting));
        else if (shifting < 0)
            dx[i] = __float2half(__half2float(dx[i]) * (1 << (-shifting)));

#if __CUDA_ARCH__ >= 530
        dx[i] = (__hgt(x[i], __hneg(threshold)) && __hlt(x[i], threshold))
            ? dx[i] : __float2half(0.0f);
#else
        dx[i] = (__half2float(x[i]) > -__half2float(threshold)
                 && __half2float(x[i]) < __half2float(threshold))
            ? dx[i] : __float2half(0.0f);
#endif
    }
}

__global__ void
cudaSSaturation_backPropagate_kernel(float* x,
                                     float* dx,
                                     unsigned int size,
                                     int shifting,
                                     float threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] /= (1 << shifting);
        else if (shifting < 0)
            dx[i] *= (1 << (-shifting));

        dx[i] *= (x[i] > -threshold && x[i] < threshold)
            ? 1.0f : 0.0f;
    }
}

__global__ void
cudaDSaturation_backPropagate_kernel(double* x,
                                     double* dx,
                                     unsigned int size,
                                     int shifting,
                                     double threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        if (shifting > 0)
            dx[i] /= (1 << shifting);
        else if (shifting < 0)
            dx[i] *= (1 << (-shifting));

        dx[i] *= (x[i] > -threshold && x[i] < threshold)
            ? 1.0 : 0.0;
    }
}

// Softplus
__global__ void cudaHSoftplus_propagate_kernel(__half* x, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        x[i] = hlog(__hadd(__float2half(1.0f), hexp(x[i])));
#else
        x[i] = __float2half(log(1.0f + exp(__half2float(x[i]))));
#endif
    }
}

__global__ void cudaSSoftplus_propagate_kernel(float* x, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        x[i] = log(1.0f + exp(x[i]));
    }
}

__global__ void cudaDSoftplus_propagate_kernel(double* x, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        x[i] = log(1.0 + exp(x[i]));
    }
}

__global__ void
cudaHSoftplus_backPropagate_kernel(__half* x, __half* dx, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        dx[i] = __hmul(dx[i], (__hsub(__float2half(1.0f), hexp(__hneg(x[i])))));
#else
        dx[i] = __float2half(__half2float(dx[i])
                             * (1.0f - exp(-__half2float(x[i]))));
#endif
    }
}

__global__ void
cudaSSoftplus_backPropagate_kernel(float* x, float* dx, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        dx[i] *= (1.0f - exp(-x[i]));
    }
}

__global__ void
cudaDSoftplus_backPropagate_kernel(double* x, double* dx, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        dx[i] *= (1.0 - exp(-x[i]));
    }
}

// Rectifier
void N2D2::cudaHRectifier_propagate(half_float::half* x,
                                    unsigned int size,
                                    half_float::half leakSlope,
                                    int shifting,
                                    half_float::half clipping)
{
    cudaHRectifier_propagate_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x),
         size,
         reinterpret_cast<__half&>(leakSlope),
         shifting,
         reinterpret_cast<__half&>(clipping));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSRectifier_propagate(float* x,
                                    unsigned int size,
                                    float leakSlope,
                                    int shifting,
                                    float clipping)
{
    cudaSRectifier_propagate_kernel<<<(size + 255) / 256, 256>>>
        (x, size, leakSlope, shifting, clipping);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDRectifier_propagate(double* x,
                                    unsigned int size,
                                    double leakSlope,
                                    int shifting,
                                    double clipping)
{
    cudaDRectifier_propagate_kernel<<<(size + 255) / 256, 256>>>
        (x, size, leakSlope, shifting, clipping);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHRectifier_backPropagate(half_float::half* x,
                                        half_float::half* dx,
                                        unsigned int size,
                                        half_float::half leakSlope,
                                        int shifting,
                                        half_float::half clipping)
{
    cudaHRectifier_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x),
         reinterpret_cast<__half*>(dx),
         size,
         reinterpret_cast<__half&>(leakSlope),
         shifting,
         reinterpret_cast<__half&>(clipping));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSRectifier_backPropagate(float* x,
                                        float* dx,
                                        unsigned int size,
                                        float leakSlope,
                                        int shifting,
                                        float clipping)
{
    cudaSRectifier_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size, leakSlope, shifting, clipping);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDRectifier_backPropagate(double* x,
                                        double* dx,
                                        unsigned int size,
                                        double leakSlope,
                                        int shifting,
                                        double clipping)
{
    cudaDRectifier_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size, leakSlope, shifting, clipping);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

// Saturation
void N2D2::cudaHSaturation_propagate(half_float::half* x,
                                     unsigned int size,
                                     int shifting,
                                     half_float::half threshold)
{
    cudaHSaturation_propagate_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x),
         size,
         shifting,
         reinterpret_cast<__half&>(threshold));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSSaturation_propagate(float* x,
                                     unsigned int size,
                                     int shifting,
                                     float threshold)
{
    cudaSSaturation_propagate_kernel<<<(size + 255) / 256, 256>>>
        (x, size, shifting, threshold);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDSaturation_propagate(double* x,
                                     unsigned int size,
                                     int shifting,
                                     double threshold)
{
    cudaDSaturation_propagate_kernel<<<(size + 255) / 256, 256>>>
        (x, size, shifting, threshold);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHSaturation_backPropagate(half_float::half* x,
                                         half_float::half* dx,
                                         unsigned int size,
                                         int shifting,
                                         half_float::half threshold)
{
    cudaHSaturation_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x),
         reinterpret_cast<__half*>(dx),
         size,
         shifting,
         reinterpret_cast<__half&>(threshold));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSSaturation_backPropagate(float* x,
                                         float* dx,
                                         unsigned int size,
                                         int shifting,
                                         float threshold)
{
    cudaSSaturation_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size, shifting, threshold);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void
N2D2::cudaDSaturation_backPropagate(double* x,
                                    double* dx,
                                    unsigned int size,
                                    int shifting,
                                    double threshold)
{
    cudaDSaturation_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size, shifting, threshold);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

// Softplus
void N2D2::cudaHSoftplus_propagate(half_float::half* x, unsigned int size)
{
    cudaHSoftplus_propagate_kernel<<<(size + 255) / 256, 256>>>(
                                            reinterpret_cast<__half*>(x), size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSSoftplus_propagate(float* x, unsigned int size)
{
    cudaSSoftplus_propagate_kernel<<<(size + 255) / 256, 256>>>(x, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDSoftplus_propagate(double* x, unsigned int size)
{
    cudaDSoftplus_propagate_kernel<<<(size + 255) / 256, 256>>>(x, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHSoftplus_backPropagate(half_float::half* x,
                                       half_float::half* dx,
                                       unsigned int size)
{
    cudaHSoftplus_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x), reinterpret_cast<__half*>(dx), size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSSoftplus_backPropagate(float* x, float* dx, unsigned int size)
{
    cudaSSoftplus_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDSoftplus_backPropagate(double* x, double* dx, unsigned int size)
{
    cudaDSoftplus_backPropagate_kernel<<<(size + 255) / 256, 256>>>
        (x, dx, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}
