#include "hip/hip_runtime.h"
/*
    (C) Copyright 2019 CEA LIST. All Rights Reserved.
    Contributor(s): Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include <cassert>
#include "CudaUtils.hpp"
#include "Scaling_CUDA_Kernels.hpp"
#include "third_party/half.hpp"

using N2D2::Float_T;
using N2D2::Cuda::clamp;

template<typename T>
__device__ T saturate(T value, std::size_t quantizedNbBits, bool isOutputUnsigned) {
    assert(quantizedNbBits > 0);

    const T min = isOutputUnsigned?0:
                                  -(1ll << (quantizedNbBits - 1ll));
    const T max = isOutputUnsigned?(1ll << quantizedNbBits) - 1ll:
                                   (1ll << (quantizedNbBits - 1ll)) - 1ll;

    return clamp(value, min, max);
}

template<typename T>
__device__ T Clip(T value, Float_T clip) {
    T res = (value < T(0.0)) ? T(0.0) : (value > T(clip)) ? T(clip) : value;
    return res;
}

template<typename T>
__device__ T Scale(T value, Float_T scale) {
    T res = value*T(scale);
    return res;
}

template<typename T>
__global__ void cudaFloatingPointScaling_kernel(const T* input, T* output,
                                                std::size_t batchSize, std::size_t nbChannels,
                                                std::size_t heigth, std::size_t width,
                                                bool isClipped,
                                                Float_T* clippingFactorPerChannel,
                                                Float_T* scalingFactorPerChannel, 
                                                std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    const std::size_t startBatch = blockIdx.x*blockDim.x + threadIdx.x;
    const std::size_t startI = blockIdx.y*blockDim.y + threadIdx.y;
    
    const std::size_t strideBatch = blockDim.x*gridDim.x;
    const std::size_t strideI = blockDim.y*gridDim.y;
    
    
    for (std::size_t batch = startBatch; batch < batchSize; batch += strideBatch) {
        for(std::size_t ch = 0; ch < nbChannels; ch++) {
            for (std::size_t i = startI; i < heigth*width; i += strideI) {
                const std::size_t index = batch*nbChannels*heigth*width + 
                                          ch*heigth*width +
                                          i;

                //clipping before scaling
                T res = isClipped ? Clip(input[index], clippingFactorPerChannel[ch]) 
                                    : input[index];
                res = Scale(res, scalingFactorPerChannel[ch]);
                if(quantizedNbBits > 0) {
                    res = saturate(round(res), quantizedNbBits, isOutputUnsigned);
                }
                output[index] = res;
            }
        }
    }
}

template<typename T>
__global__ void cudaFixedPointScaling_kernel(const T* input, T* output,
                                             std::size_t batchSize, std::size_t nbChannels,
                                             std::size_t heigth, std::size_t width,
                                             bool isClipped, Float_T* clippingFactorPerChannel,
                                             std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                             std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    assert(quantizedNbBits > 0);
        
    const std::size_t startBatch = blockIdx.x*blockDim.x + threadIdx.x;
    const std::size_t startI = blockIdx.y*blockDim.y + threadIdx.y;
    
    const std::size_t strideBatch = blockDim.x*gridDim.x;
    const std::size_t strideI = blockDim.y*gridDim.y;
    
    
    for (std::size_t batch = startBatch; batch < batchSize; batch += strideBatch) {
        for(std::size_t ch = 0; ch < nbChannels; ch++) {
            for (std::size_t i = startI; i < heigth*width; i += strideI) {
                const std::size_t index = batch*nbChannels*heigth*width + 
                                          ch*heigth*width +
                                          i;
                
                T realInput = isClipped ? Clip(input[index], clippingFactorPerChannel[ch]) 
                                    : input[index]; 

                const long long half = (nbFractionalBits > 0)
                    ? (1ll << (nbFractionalBits - 1))
                    : 0ll;

                long long rInput = round(realInput);
                const long long res = (
                    static_cast<long long>(rInput) * scalingFactorPerChannel[ch] + half
                )  >> nbFractionalBits;
                

                output[index] = saturate(res, quantizedNbBits, isOutputUnsigned);
            }
        }
    }
}

template<typename T>
__global__ void cudaSingleShiftScaling_kernel(const T* input, T* output,
                                                std::size_t batchSize, std::size_t nbChannels,
                                                std::size_t heigth, std::size_t width,
                                                bool isClipped, Float_T* clippingFactorPerChannel,
                                                unsigned char* scalingFactorPerChannel,
                                                std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    const std::size_t startBatch = blockIdx.x*blockDim.x + threadIdx.x;
    const std::size_t startI = blockIdx.y*blockDim.y + threadIdx.y;
    
    const std::size_t strideBatch = blockDim.x*gridDim.x;
    const std::size_t strideI = blockDim.y*gridDim.y;
    
    
    for (std::size_t batch = startBatch; batch < batchSize; batch += strideBatch) {
        for(std::size_t ch = 0; ch < nbChannels; ch++) {
            for (std::size_t i = startI; i < heigth*width; i += strideI) {
                const std::size_t index = batch*nbChannels*heigth*width + 
                                          ch*heigth*width +
                                          i;
                
                //TODO::add clipping here properly, nothing for now
                /*
                const long long half = (scalingFactorPerChannel[ch] > 0)
                    ? (1ll << (scalingFactorPerChannel[ch] - 1))
                    : 0ll;
                const long long res = (
                    static_cast<long long>(round(input[index])) + half
                ) >> scalingFactorPerChannel[ch];
                */
                T realInput = input[index];
                if(isClipped){
                    realInput = (realInput > T(clippingFactorPerChannel[ch])) ? T(clippingFactorPerChannel[ch]) : realInput;
                }

                const long long half = (scalingFactorPerChannel[ch] > 0)
                ? (1ll << (scalingFactorPerChannel[ch] - 1))
                : 0ll;

                long long rInput = round(realInput);

                const long long res = (
                    static_cast<long long>(rInput) + half
                ) >> scalingFactorPerChannel[ch];

                output[index] = saturate(res, quantizedNbBits, isOutputUnsigned);
            }
        }
    }
}

template<typename T>
__global__ void cudaDoubleShiftScaling_kernel(const T* input, T* output,
                                              std::size_t batchSize, std::size_t nbChannels,
                                              std::size_t heigth, std::size_t width,
                                              bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                              std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                              std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    const std::size_t startBatch = blockIdx.x*blockDim.x + threadIdx.x;
    const std::size_t startI = blockIdx.y*blockDim.y + threadIdx.y;
    
    const std::size_t strideBatch = blockDim.x*gridDim.x;
    const std::size_t strideI = blockDim.y*gridDim.y;
    
    
    for (std::size_t batch = startBatch; batch < batchSize; batch += strideBatch) {
        for(std::size_t ch = 0; ch < nbChannels; ch++) {
            for (std::size_t i = startI; i < heigth*width; i += strideI) {
                const std::size_t index = batch*nbChannels*heigth*width + 
                                          ch*heigth*width +
                                          i;

                //TODO::add clipping here properly, nothing for now  
                const long long half = (scalingFactorPerChannel[ch].second > 0)
                    ? (1ll << (scalingFactorPerChannel[ch].second - 1))
                    : 0ll;
                const long long val = static_cast<long long>(round(input[index]));
                const long long res = (
                    val + (val << scalingFactorPerChannel[ch].first) +  half
                ) >> scalingFactorPerChannel[ch].second;
                
                /*
                const long long half = (scalingFactorPerChannel[ch].second > 0)
                ? (1ll << (scalingFactorPerChannel[ch].second - 1))
                : 0ll;

                long long val = round(input[index]);
                if(isClipped){
                    val = (val > clippingFactorPerChannel[ch]) ? clippingFactorPerChannel[ch] : val;
                }

                const long long res = (
                    val + (val << scalingFactorPerChannel[ch].first) +  half
                ) >> scalingFactorPerChannel[ch].second;
                */

                output[index] = saturate(res, quantizedNbBits, isOutputUnsigned);
            }
        }
    }
}





namespace N2D2 {

template<>
void cudaFloatingPointScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                                const half_float::half* input, half_float::half* output,
                                                                std::size_t batchSize, std::size_t nbChannels,
                                                                std::size_t heigth, std::size_t width,
                                                                bool isClipped,
                                                                Float_T* clippingFactorPerChannel,
                                                                Float_T* scalingFactorPerChannel,
                                                                std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    throw std::runtime_error("Floating-point scaling cell doesn't support half-floats.");
}


template<typename T>
void cudaFloatingPointScaling_propagate(const hipDeviceProp_t& deviceProp,
                                              const T* input, T* output,
                                              std::size_t batchSize, std::size_t nbChannels,
                                              std::size_t heigth, std::size_t width,
                                              bool isClipped,
                                              Float_T* clippingFactorPerChannel,
                                              Float_T* scalingFactorPerChannel,
                                              std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    // TODO Optimize dimensions based on the size of the batch and cell
    const dim3 threadsPerBlock = dim3(deviceProp.maxThreadsPerBlock/deviceProp.warpSize, 
                            deviceProp.warpSize);
    const dim3 blocksPerGrid = dim3(16, deviceProp.multiProcessorCount);

    cudaFloatingPointScaling_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, 
                                                                        batchSize, nbChannels, 
                                                                        heigth, width, 
                                                                        isClipped,
                                                                        clippingFactorPerChannel,
                                                                        scalingFactorPerChannel,
                                                                        quantizedNbBits, isOutputUnsigned);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}






template<>
void cudaFixedPointScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                             const half_float::half* input, half_float::half* output,
                                                             std::size_t batchSize, std::size_t nbChannels,
                                                             std::size_t heigth, std::size_t width,
                                                             bool isClipped, Float_T* clippingFactorPerChannel,
                                                             std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                                             std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    throw std::runtime_error("Fixed-point scaling cell doesn't support half-floats.");
}

template<typename T>
void cudaFixedPointScaling_propagate(const hipDeviceProp_t& deviceProp,
                                           const T* input, T* output,
                                           std::size_t batchSize, std::size_t nbChannels,
                                           std::size_t heigth, std::size_t width,
                                           bool isClipped, Float_T* clippingFactorPerChannel,
                                           std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                           std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    // TODO Optimize dimensions based on the size of the batch and cell
    const dim3 threadsPerBlock = dim3(deviceProp.maxThreadsPerBlock/deviceProp.warpSize, 
                            deviceProp.warpSize);
    const dim3 blocksPerGrid = dim3(16, deviceProp.multiProcessorCount);

    cudaFixedPointScaling_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, 
                                                                     batchSize, nbChannels, 
                                                                     heigth, width, 
                                                                     isClipped, clippingFactorPerChannel,
                                                                     scalingFactorPerChannel, nbFractionalBits,
                                                                     quantizedNbBits, isOutputUnsigned);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}







template<>
void cudaSingleShiftScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                              const half_float::half* input, half_float::half* output,
                                                              std::size_t batchSize, std::size_t nbChannels,
                                                              std::size_t heigth, std::size_t width,
                                                              bool isClipped, Float_T* clippingFactorPerChannel,
                                                              unsigned char* scalingFactorPerChannel,
                                                              std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    throw std::runtime_error("Single-shift scaling cell doesn't support half-floats.");
}

template<typename T>
void cudaSingleShiftScaling_propagate(const hipDeviceProp_t& deviceProp,
                                            const T* input, T* output,
                                            std::size_t batchSize, std::size_t nbChannels,
                                            std::size_t heigth, std::size_t width,
                                            bool isClipped, Float_T* clippingFactorPerChannel,
                                            unsigned char* scalingFactorPerChannel,
                                            std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    // TODO Optimize dimensions based on the size of the batch and cell
    const dim3 threadsPerBlock = dim3(deviceProp.maxThreadsPerBlock/deviceProp.warpSize, 
                            deviceProp.warpSize);
    const dim3 blocksPerGrid = dim3(16, deviceProp.multiProcessorCount);

    cudaSingleShiftScaling_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, 
                                                                      batchSize, nbChannels, 
                                                                      heigth, width, 
                                                                      isClipped, clippingFactorPerChannel,
                                                                      scalingFactorPerChannel,
                                                                      quantizedNbBits, isOutputUnsigned);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}






template<>
void cudaDoubleShiftScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                              const half_float::half* input, half_float::half* output,
                                                              std::size_t batchSize, std::size_t nbChannels,
                                                              std::size_t heigth, std::size_t width,
                                                              bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                                              std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                                              std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    throw std::runtime_error("Double-shift scaling cell doesn't support half-floats.");
}

template<typename T>
void cudaDoubleShiftScaling_propagate(const hipDeviceProp_t& deviceProp,
                                            const T* input, T* output,
                                            std::size_t batchSize, std::size_t nbChannels,
                                            std::size_t heigth, std::size_t width,
                                            bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                            std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                            std::size_t quantizedNbBits, bool isOutputUnsigned)
{
    // TODO Optimize dimensions based on the size of the batch and cell
    const dim3 threadsPerBlock = dim3(deviceProp.maxThreadsPerBlock/deviceProp.warpSize, 
                            deviceProp.warpSize);
    const dim3 blocksPerGrid = dim3(16, deviceProp.multiProcessorCount);

    cudaDoubleShiftScaling_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, 
                                                                      batchSize, nbChannels, 
                                                                      heigth, width, 
                                                                      isClipped, clippingFactorPerChannel,
                                                                      scalingFactorPerChannel,
                                                                      quantizedNbBits, isOutputUnsigned);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}






template void cudaFloatingPointScaling_propagate<float>(const hipDeviceProp_t& deviceProp,
                                                              const float* input, float* output,
                                                              std::size_t batchSize, std::size_t nbChannels,
                                                              std::size_t heigth, std::size_t width,
                                                              bool isClipped, Float_T* clippingFactorPerChannel,
                                                              Float_T* scalingFactorPerChannel,
                                                              std::size_t quantizedNbBits, bool isOutputUnsigned);

template void cudaFloatingPointScaling_propagate<double>(const hipDeviceProp_t& deviceProp,
                                                               const double* input, double* output,
                                                               std::size_t batchSize, std::size_t nbChannels,
                                                               std::size_t heigth, std::size_t width,
                                                               bool isClipped, Float_T* clippingFactorPerChannel,
                                                               Float_T* scalingFactorPerChannel,
                                                               std::size_t quantizedNbBits, bool isOutputUnsigned);

template void cudaFloatingPointScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                                         const half_float::half* input, half_float::half* output,
                                                                         std::size_t batchSize, std::size_t nbChannels,
                                                                         std::size_t heigth, std::size_t width,
                                                                         bool isClipped, Float_T* clippingFactorPerChannel,
                                                                         Float_T* scalingFactorPerChannel,
                                                                         std::size_t quantizedNbBits, bool isOutputUnsigned);


template void cudaFixedPointScaling_propagate<float>(const hipDeviceProp_t& deviceProp,
                                                           const float* input, float* output,
                                                           std::size_t batchSize, std::size_t nbChannels,
                                                           std::size_t heigth, std::size_t width,
                                                           bool isClipped, Float_T* clippingFactorPerChannel,
                                                           std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                                           std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaFixedPointScaling_propagate<double>(const hipDeviceProp_t& deviceProp,
                                                            const double* input, double* output,
                                                            std::size_t batchSize, std::size_t nbChannels,
                                                            std::size_t heigth, std::size_t width,
                                                            bool isClipped, Float_T* clippingFactorPerChannel,
                                                            std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                                            std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaFixedPointScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                                      const half_float::half* input, half_float::half* output,
                                                                      std::size_t batchSize, std::size_t nbChannels,
                                                                      std::size_t heigth, std::size_t width,
                                                                      bool isClipped, Float_T* clippingFactorPerChannel,
                                                                      std::int32_t* scalingFactorPerChannel, std::size_t nbFractionalBits,
                                                                      std::size_t quantizedNbBits, bool isOutputUnsigned);


template void cudaSingleShiftScaling_propagate<float>(const hipDeviceProp_t& deviceProp,
                                                            const float* input, float* output,
                                                            std::size_t batchSize, std::size_t nbChannels,
                                                            std::size_t heigth, std::size_t width,
                                                            bool isClipped, Float_T* clippingFactorPerChannel,
                                                            unsigned char* scalingFactorPerChannel,
                                                            std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaSingleShiftScaling_propagate<double>(const hipDeviceProp_t& deviceProp,
                                                             const double* input, double* output,
                                                             std::size_t batchSize, std::size_t nbChannels,
                                                             std::size_t heigth, std::size_t width,
                                                             bool isClipped, Float_T* clippingFactorPerChannel,
                                                             unsigned char* scalingFactorPerChannel,
                                                             std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaSingleShiftScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                                       const half_float::half* input, half_float::half* output,
                                                                       std::size_t batchSize, std::size_t nbChannels,
                                                                       std::size_t heigth, std::size_t width,
                                                                       bool isClipped, Float_T* clippingFactorPerChannel,
                                                                       unsigned char* scalingFactorPerChannel,
                                                                       std::size_t quantizedNbBits, bool isOutputUnsigned);


template void cudaDoubleShiftScaling_propagate<float>(const hipDeviceProp_t& deviceProp,
                                                            const float* input, float* output,
                                                            std::size_t batchSize, std::size_t nbChannels,
                                                            std::size_t heigth, std::size_t width,
                                                            bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                                            std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                                            std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaDoubleShiftScaling_propagate<double>(const hipDeviceProp_t& deviceProp,
                                                             const double* input, double* output,
                                                             std::size_t batchSize, std::size_t nbChannels,
                                                             std::size_t heigth, std::size_t width,
                                                             bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                                             std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                                             std::size_t quantizedNbBits, bool isOutputUnsigned);
template void cudaDoubleShiftScaling_propagate<half_float::half>(const hipDeviceProp_t& deviceProp,
                                                                       const half_float::half* input, half_float::half* output,
                                                                       std::size_t batchSize, std::size_t nbChannels,
                                                                       std::size_t heigth, std::size_t width,
                                                                       bool isClipped, std::pair<unsigned char, unsigned char>* clippingFactorPerChannel,
                                                                       std::pair<unsigned char, unsigned char>* scalingFactorPerChannel,
                                                                       std::size_t quantizedNbBits, bool isOutputUnsigned);
}