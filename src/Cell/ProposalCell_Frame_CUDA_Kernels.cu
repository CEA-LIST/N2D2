#include "hip/hip_runtime.h"

/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): David BRIAND(david.briand@cea.fr)
                    Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include "Cell/ProposalCell_Frame_CUDA_Kernels.hpp"

__global__ void cudaSNormalizeROIs_kernel( unsigned int inputSizeX,
                                            unsigned int inputSizeY,
                                            unsigned int nbProposals,
                                            unsigned int batchSize,
                                            unsigned int scoreIdx,
                                            unsigned int nbCls,
                                            bool keepMax,
                                            const float normX,
                                            const float normY,
                                            const float* means,
                                            const float* std,
                                            const float* ROIRef,
                                            float* ROIEst,
                                            float* ValueEst,
                                            float* outputs,
                                            float* argMax,
                                            float scoreThreshold)
{
    const int batchPos = blockIdx.z*nbProposals;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

    if(index < nbProposals)
    {
        unsigned int indexMin = scoreIdx;
        unsigned int indexMax = nbCls;

        if(keepMax)
        {
            unsigned int cls = scoreIdx;
            float maxVal = 0.0;

            for(unsigned int i = indexMin; i < indexMax; ++i)
            {
                unsigned int inputIdx = i + index*nbCls + batchPos*nbCls;

                if (ValueEst[inputIdx] >= maxVal)
                {
                    maxVal = ValueEst[inputIdx];
                    cls = i;
                }

            }
            argMax[index + batchPos] = cls;

            indexMin = cls;
            indexMax = cls + 1;
        }

        for(unsigned int clsIdx = indexMin; clsIdx < indexMax; ++clsIdx)
        {

            unsigned int bboxRefIdx = index*4 + batchPos*4;
            unsigned int bboxEstIdx = clsIdx*4 + index*4*nbCls + batchPos*4*nbCls;
            unsigned int valEstIdx = clsIdx + index*nbCls + batchPos*nbCls;
            //unsigned int outputIdx = keepMax ? index*4 + batchPos*4 : 
            //                            (clsIdx - scoreIdx)*4 + index*4*nbCls + batchPos*4*nbCls;
            unsigned int outputIdx = keepMax ? index*4*(nbCls - scoreIdx) + batchPos*4*(nbCls - scoreIdx)
                                        : (clsIdx - scoreIdx)*4 + index*4*(nbCls - scoreIdx) + batchPos*4*(nbCls - scoreIdx);


            const float xbbRef = ROIRef[0 + bboxRefIdx]*normX;
            const float ybbRef = ROIRef[1 + bboxRefIdx]*normY;
            const float wbbRef = ROIRef[2 + bboxRefIdx]*normX;
            const float hbbRef = ROIRef[3 + bboxRefIdx]*normY;


            const float xbbEst = ROIEst[0 + bboxEstIdx]*std[0] + means[0];

            const float ybbEst = ROIEst[1 + bboxEstIdx]*std[1] + means[1];

            const float wbbEst = ROIEst[2 + bboxEstIdx]*std[2] + means[2];

            const float hbbEst = ROIEst[3 + bboxEstIdx]*std[3] + means[3];


            float x = xbbEst*wbbRef + xbbRef + wbbRef/2.0 - (wbbRef/2.0)*exp(wbbEst);
            float y = ybbEst*hbbRef + ybbRef + hbbRef/2.0 - (hbbRef/2.0)*exp(hbbEst);
            float w = wbbRef*exp(wbbEst);
            float h = hbbRef*exp(hbbEst);

            /**Clip values**/
            if(x < 0.0)
            {
                w += x;
                x = 0.0;
            }

            if(y < 0.0)
            {
                h += y;
                y = 0.0;
            }

            w = ((w + x) > 1.0) ? (1.0 - x) / normX : w / normX;
            h = ((h + y) > 1.0) ? (1.0 - y) / normY : h / normY;

            x /= normX;
            y /= normY;
            
            if(ValueEst[valEstIdx] >= scoreThreshold)
            {
                outputs[0 + outputIdx] = x;
                outputs[1 + outputIdx] = y;
                outputs[2 + outputIdx] = w;
                outputs[3 + outputIdx] = h;

            }
            else
            {
                outputs[0 + outputIdx] = 0.0;
                outputs[1 + outputIdx] = 0.0;
                outputs[2 + outputIdx] = 0.0;
                outputs[3 + outputIdx] = 0.0;
            }    
        }
    }

}


__global__ void cudaSToOutput_kernel( const unsigned int nbProposals,
                                      const unsigned int scoreIdx,
                                      const unsigned int nbCls,
                                      const float* ROIEst,
                                      float* outputs)
{
    const int batchPos = blockIdx.z*nbProposals;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

    if(index < nbProposals)
    {
        const unsigned int inputIdx = index*4*(nbCls - scoreIdx) 
                                            + batchPos*4*(nbCls - scoreIdx);

        outputs[0 + index*4 + batchPos*4] = ROIEst[0 + inputIdx];
        outputs[1 + index*4 + batchPos*4] = ROIEst[1 + inputIdx];
        outputs[2 + index*4 + batchPos*4] = ROIEst[2 + inputIdx];
        outputs[3 + index*4 + batchPos*4] = ROIEst[3 + inputIdx];        
    }

}
void N2D2::cudaSNormalizeROIs(unsigned int inputSizeX,
                        unsigned int inputSizeY,
                        unsigned int nbProposals,
                        unsigned int batchSize,
                        unsigned int scoreIdx,
                        unsigned int nbCls,
                        bool keepMax,
                        const float normX,
                        const float normY,
                        const float* means,
                        const float* std,
                        const float* ROIRef,
                        float* ROIEst,
                        float* ValueEst,
                        float* outputs,
                        float* argMax,
                        float scoreThreshold,
                        const dim3 threadsPerBlock,
                        const dim3 blocksPerGrid)
{

    cudaSNormalizeROIs_kernel<<<blocksPerGrid, threadsPerBlock>>>( inputSizeX,
                                                                    inputSizeY, 
                                                                    nbProposals,
                                                                    batchSize, 
                                                                    scoreIdx,
                                                                    nbCls,
                                                                    keepMax,
                                                                    normX, 
                                                                    normY, 
                                                                    means, 
                                                                    std, 
                                                                    ROIRef, 
                                                                    ROIEst,
                                                                    ValueEst,
                                                                    outputs,
                                                                    argMax,
                                                                    scoreThreshold);
    CHECK_CUDA_STATUS(hipPeekAtLastError());

}


void N2D2::cudaSToOutputROIs(const unsigned int nbProposals,
                             const unsigned int scoreIdx,
                             const unsigned int nbCls,
                             const float* ROIEst,
                             float* outputs,
                             const dim3 threadsPerBlock,
                             const dim3 blocksPerGrid)
{

    cudaSToOutput_kernel<<<blocksPerGrid, threadsPerBlock>>>( nbProposals,
                                                              scoreIdx,
                                                              nbCls,
                                                              ROIEst, 
                                                              outputs);
    CHECK_CUDA_STATUS(hipPeekAtLastError());

}
