#include "hip/hip_runtime.h"
/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include "Cell/FMPCell_Frame_CUDA_Kernels.hpp"

__global__ void cudaSFMPPropagate_kernel(float* inputs,
                                         unsigned int* gridX,
                                         unsigned int* gridY,
                                         float* outputs,
                                         unsigned int nbChannels,
                                         unsigned int channelsHeight,
                                         unsigned int channelsWidth,
                                         unsigned int nbOutputs,
                                         unsigned int outputsHeight,
                                         unsigned int outputsWidth,
                                         unsigned int batchSize,
                                         bool overlapping)
{
    const unsigned int batchInputOffset = blockIdx.z * nbChannels
                                          * channelsHeight * channelsWidth;
    const unsigned int batchOutputOffset = blockIdx.z * nbOutputs
                                           * outputsHeight * outputsWidth;

    for (unsigned int output = blockIdx.x; output < nbOutputs;
         output += gridDim.x) {
        for (unsigned int oy = threadIdx.y; oy < outputsHeight;
             oy += blockDim.y) {
            for (unsigned int ox = threadIdx.x; ox < outputsWidth;
                 ox += blockDim.x) {
                // For each output, compute the pool value
                float poolValue = -FLT_MAX;
                /*
                                unsigned int channelMax = 0;
                                unsigned int ixMax = 0;
                                unsigned int iyMax = 0;
                */
                const unsigned int ixStart = (ox > 0) ? gridX[ox - 1] : 0;
                const unsigned int iyStart = (oy > 0) ? gridY[oy - 1] : 0;
                unsigned int ixStop = gridX[ox];
                unsigned int iyStop = gridY[oy];

                if (!overlapping) {
                    --ixStop;
                    --iyStop;
                }

                if (ox == outputsWidth - 1)
                    ixStop = channelsWidth - 1;

                if (oy == outputsHeight - 1)
                    iyStop = channelsHeight - 1;

                for (unsigned int iy = iyStart; iy <= iyStop; ++iy) {
                    for (unsigned int ix = ixStart; ix <= ixStop; ++ix) {
                        const unsigned int inputsIdx
                            = ix + (iy + output * channelsHeight)
                                   * channelsWidth;

                        if (inputs[inputsIdx + batchInputOffset] > poolValue) {
                            poolValue = inputs[inputsIdx + batchInputOffset];
                            /*
                                                        channelMax = channel;
                                                        ixMax = ix;
                                                        iyMax = iy;
                            */
                        }
                    }
                }

                // Compute the output signal
                const unsigned int outputsIdx
                    = ox + (oy + output * outputsHeight) * outputsWidth;
                outputs[outputsIdx + batchOutputOffset] = poolValue;
            }
        }
    }
}

static unsigned int nextDivisor(unsigned int target, unsigned int value)
{
    unsigned int v = value;
    while (target % v != 0)
        ++v;
    return v;
}

void N2D2::cudaSFMPPropagate(float* inputs,
                             unsigned int* gridX,
                             unsigned int* gridY,
                             float* outputs,
                             unsigned int nbChannels,
                             unsigned int channelsHeight,
                             unsigned int channelsWidth,
                             unsigned int nbOutputs,
                             unsigned int outputsHeight,
                             unsigned int outputsWidth,
                             unsigned int batchSize,
                             bool overlapping)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const unsigned int maxSize = (unsigned int)deviceProp.maxThreadsPerBlock;
    const unsigned int prefMultiple = (unsigned int)deviceProp.warpSize;

    const unsigned int groupSize = (outputsWidth * outputsHeight < maxSize)
                                       ? outputsWidth * outputsHeight
                                       : maxSize;
    const unsigned int groupWidth
        = min(prefMultiple, nextDivisor(groupSize, outputsWidth));

    const dim3 blocksPerGrid = {nbOutputs, 1, batchSize};
    const dim3 threadsPerBlocks = {groupWidth, groupSize / groupWidth, 1};

    cudaSFMPPropagate_kernel<<<blocksPerGrid, threadsPerBlocks>>>
        (inputs,
           gridX,
           gridY,
           outputs,
           nbChannels,
           channelsHeight,
           channelsWidth,
           nbOutputs,
           outputsHeight,
           outputsWidth,
           batchSize,
           overlapping);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}
