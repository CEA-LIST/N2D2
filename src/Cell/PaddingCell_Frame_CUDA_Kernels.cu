#include "hip/hip_runtime.h"

/*
    (C) Copyright 2018 CEA LIST. All Rights Reserved.
    Contributor(s): David BRIAND(david.briand@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include "Cell/PaddingCell_Frame_CUDA_Kernels.hpp"

__global__ void cudaSPadding_kernel( unsigned int outputWidth,
                                     unsigned int outputHeight,
                                     unsigned int nbChannels,
                                     unsigned int batchSize,
                                     unsigned int inputWidth,
                                     unsigned int inputHeight,
                                     int leftPad,
                                     int rightPad,
                                     int topPad,
                                     int botPad,
                                     const float* input,
                                     float* outputs)
{

    const unsigned int inputOffset
        = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*inputWidth*inputHeight; 

    const unsigned int outputOffset
        = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels*outputWidth*outputHeight;

    for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x) 
    {
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) 
        {
            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) 
            {
                float outputValue = 0.0;
                int ix = (int) ox - leftPad;
                int iy = (int) oy - topPad;

                if( ix >= 0 && ix < (int) inputWidth
                    && iy >= 0 && iy < (int) inputHeight )
                {
                    outputValue = input[ix +  
                                        iy*inputWidth 
                                        + ch*inputWidth*inputHeight
                                        + inputOffset];

                }
                outputs[ ox + oy*outputWidth 
                         + ch*outputWidth*outputHeight + outputOffset]  = outputValue;

            }
        }
    }


}

void N2D2::cudaSPadding(unsigned int outputSizeX,
                        unsigned int outputSizeY,
                        unsigned int outputNbChannels,
                        unsigned int batchSize,
                        unsigned int inputSizeX,
                        unsigned int inputSizeY,
                        int paddingLeft,
                        int paddingRight,
                        int paddingTop,
                        int paddingBot,
                        const float* input,
                        float* outputs)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const unsigned int maxSize = (unsigned int)deviceProp.maxThreadsPerBlock;
    const unsigned int prefMultiple = (unsigned int)deviceProp.warpSize;

    const unsigned int groupSize = (outputSizeX * outputSizeY < maxSize)
                                       ? outputSizeX * outputSizeY
                                       : maxSize;
    const unsigned int reqWidth = (unsigned int) ceil((float) groupSize / (float) outputSizeX);

    const unsigned int groupWidth = min(prefMultiple, reqWidth);

    const dim3 blocksPerGrid = {outputNbChannels, 1, batchSize};
    const dim3 threadsPerBlock = {groupWidth, groupSize / groupWidth, 1};

    cudaSPadding_kernel<<<blocksPerGrid, threadsPerBlock>>>( outputSizeX,
                                                            outputSizeY, 
                                                            outputNbChannels,
                                                            batchSize, 
                                                            inputSizeX,
                                                            inputSizeY, 
                                                            paddingLeft, 
                                                            paddingRight, 
                                                            paddingTop, 
                                                            paddingBot, 
                                                            input,
                                                            outputs);
    CHECK_CUDA_STATUS(hipPeekAtLastError());

}
