#include "hip/hip_runtime.h"
/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include "Cell/NormalizeCell_Frame_CUDA_Kernels.hpp"

////Forward
template <class T>
__global__
void cudaNormalizeL2Forward_kernel(const T alpha,
                                    T* inputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth,
                                    unsigned int batchSize,
                                    const T beta,
                                    T* outputs,
                                    T* normData,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth)
{
    const unsigned int batchOffset = blockIdx.z * nbOutputs
                                           * outputsHeight * outputsWidth;

    for (unsigned int oy = threadIdx.y; oy < outputsHeight;
            oy += blockDim.y) {
        for (unsigned int ox = threadIdx.x; ox < outputsWidth;
                ox += blockDim.x)
        {
            T sumSq = 0.0f;

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                sumSq += inputs[idx] * inputs[idx];
            }

            const T scale = sqrt(sumSq + 1.0e-6);

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                normData[idx] = scale;
                outputs[idx] = inputs[idx] / scale;
            }
        }
    }
}

template <>
__global__
void cudaNormalizeL2Forward_kernel<__half>(const __half alpha,
                                    __half* inputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth,
                                    unsigned int batchSize,
                                    const __half beta,
                                    __half* outputs,
                                    __half* normData,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth)
{
    const unsigned int batchOffset = blockIdx.z * nbOutputs
                                           * outputsHeight * outputsWidth;

    for (unsigned int oy = threadIdx.y; oy < outputsHeight;
            oy += blockDim.y) {
        for (unsigned int ox = threadIdx.x; ox < outputsWidth;
                ox += blockDim.x)
        {
#if __CUDA_ARCH__ >= 530
            __half sumSq = __float2half(0.0f);

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                sumSq = __hadd(sumSq, __hmul(inputs[idx], inputs[idx]));
            }

            const __half scale = __float2half(
                sqrt(__half2float(sumSq) + 1.0e-6));

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                normData[idx] = scale;
                outputs[idx] = __float2half(__half2float(inputs[idx])
                                    / __half2float(scale));
            }
#else
            float sumSq(0.0f);

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                sumSq += __half2float(inputs[idx]) * __half2float(inputs[idx]);
            }

            const float scale = sqrt(sumSq + 1.0e-6);

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                normData[idx] = __float2half(scale);
                outputs[idx] = __float2half(__half2float(inputs[idx]) / scale);
            }
#endif
        }
    }
}

// Backward
template <class T>
__global__
void cudaNormalizeL2Backward_kernel(const T alpha,
                                     T* outputs,
                                     T* normData,
                                     T* diffInputs,
                                     unsigned int nbOutputs,
                                     unsigned int outputsHeight,
                                     unsigned int outputsWidth,
                                     unsigned int batchSize,
                                     const T beta,
                                     T* diffOutputs,
                                     unsigned int nbChannels,
                                     unsigned int channelsHeight,
                                     unsigned int channelsWidth)
{
    const unsigned int batchOffset = blockIdx.z * nbOutputs
                                           * outputsHeight * outputsWidth;

    for (unsigned int oy = threadIdx.y; oy < outputsHeight;
            oy += blockDim.y) {
        for (unsigned int ox = threadIdx.x; ox < outputsWidth;
                ox += blockDim.x)
        {
            T a = 0.0f;

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                a += outputs[idx] * diffInputs[idx];
            }

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                if (beta != 0.0f) {
                    diffOutputs[idx] = (diffInputs[idx] - outputs[idx] * a)
                        / normData[idx] + beta * diffOutputs[idx];
                }
                else {
                    diffOutputs[idx] = (diffInputs[idx] - outputs[idx] * a)
                        / normData[idx];
                }
            }
        }
    }
}

template <>
__global__
void cudaNormalizeL2Backward_kernel<__half>(const __half alpha,
                                     __half* outputs,
                                     __half* normData,
                                     __half* diffInputs,
                                     unsigned int nbOutputs,
                                     unsigned int outputsHeight,
                                     unsigned int outputsWidth,
                                     unsigned int batchSize,
                                     const __half beta,
                                     __half* diffOutputs,
                                     unsigned int nbChannels,
                                     unsigned int channelsHeight,
                                     unsigned int channelsWidth)
{
    const unsigned int batchOffset = blockIdx.z * nbOutputs
                                           * outputsHeight * outputsWidth;

    for (unsigned int oy = threadIdx.y; oy < outputsHeight;
            oy += blockDim.y) {
        for (unsigned int ox = threadIdx.x; ox < outputsWidth;
                ox += blockDim.x)
        {
#if __CUDA_ARCH__ >= 530
            __half a = __float2half(0.0f);

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                a = __hadd(a, __hmul(outputs[idx], diffInputs[idx]));
            }

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                const __half factor = __float2half(1.0f
                                            / __half2float(normData[idx]));

                if (! __heq(beta, __float2half(0.0f))) {
                    diffOutputs[idx] = __hadd(
                        __hmul(__hsub(diffInputs[idx], __hmul(outputs[idx], a)),
                              factor),
                        __hmul(beta, diffOutputs[idx]));
                }
                else {
                    diffOutputs[idx] =
                        __hmul(__hsub(diffInputs[idx], __hmul(outputs[idx], a)),
                              factor);
                }
            }
#else
            float a = 0.0f;

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                a += __half2float(outputs[idx]) * __half2float(diffInputs[idx]);
            }

            for (unsigned int output = 0; output < nbOutputs; ++output) {
                const unsigned int idx = batchOffset
                    + ox + (oy + output * outputsHeight) * outputsWidth;

                if (__half2float(beta) != 0.0f) {
                    diffOutputs[idx] = __float2half(
                        (__half2float(diffInputs[idx])
                            - __half2float(outputs[idx]) * a)
                                / __half2float(normData[idx])
                        + __half2float(beta) * __half2float(diffOutputs[idx]));
                }
                else {
                    diffOutputs[idx] = __float2half(
                        (__half2float(diffInputs[idx])
                            - __half2float(outputs[idx]) * a)
                                / __half2float(normData[idx]));
                }
            }
#endif
        }
    }
}

namespace N2D2 {

template <class T>
void cudaNormalizeL2Forward(const hipDeviceProp_t& deviceProp,
                                   T alpha,
                                   T* inputs,
                                   unsigned int nbChannels,
                                   unsigned int channelsHeight,
                                   unsigned int channelsWidth,
                                   unsigned int batchSize,
                                   T beta,
                                   T* outputs,
                                   T* normData,
                                   unsigned int nbOutputs,
                                   unsigned int outputsHeight,
                                   unsigned int outputsWidth)
{
    const unsigned int maxSize = (unsigned int)deviceProp.maxThreadsPerBlock;
    const unsigned int prefMultiple = (unsigned int)deviceProp.warpSize;

    const unsigned int groupSize = (outputsWidth * outputsHeight < maxSize)
                                       ? outputsWidth * outputsHeight
                                       : maxSize;

    const unsigned int reqWidth
        = (unsigned int)ceilf((float)groupSize / (float)outputsWidth);

    const unsigned int groupWidth = min(prefMultiple, reqWidth);
    const dim3 blocksPerGrid = {1, 1, batchSize};
    const dim3 threadsPerBlocks = {groupWidth, groupSize / groupWidth, 1};

    cudaNormalizeL2Forward_kernel<<<blocksPerGrid, threadsPerBlocks>>>
        (reinterpret_cast<typename Cuda::cuda_type<T>::type&>(alpha),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(inputs),
           nbChannels,
           channelsHeight,
           channelsWidth,
           batchSize,
           reinterpret_cast<typename Cuda::cuda_type<T>::type&>(beta),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(outputs),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(normData),
           nbOutputs,
           outputsHeight,
           outputsWidth);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

template <class T>
void cudaNormalizeL2Backward(const hipDeviceProp_t& deviceProp,
                                    T alpha,
                                    T* outputs,
                                    T* normData,
                                    T* diffInputs,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth,
                                    unsigned int batchSize,
                                    T beta,
                                    T* diffOutputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth)
{
    const unsigned int maxSize = (unsigned int)deviceProp.maxThreadsPerBlock;
    const unsigned int prefMultiple = (unsigned int)deviceProp.warpSize;

    const unsigned int groupSize = (channelsWidth * channelsHeight < maxSize)
                                       ? channelsWidth * channelsHeight
                                       : maxSize;
    const unsigned int reqWidth
        = (unsigned int)ceilf((float)groupSize / (float)outputsWidth);

    const unsigned int groupWidth = min(prefMultiple, reqWidth);

    const dim3 blocksPerGrid = {1, 1, batchSize};
    const dim3 threadsPerBlocks = {groupWidth, groupSize / groupWidth, 1};

    cudaNormalizeL2Backward_kernel<<<blocksPerGrid, threadsPerBlocks>>>
        (reinterpret_cast<typename Cuda::cuda_type<T>::type&>(alpha),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(outputs),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(normData),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(diffInputs),
           nbOutputs,
           outputsHeight,
           outputsWidth,
           batchSize,
           reinterpret_cast<typename Cuda::cuda_type<T>::type&>(beta),
           reinterpret_cast<typename Cuda::cuda_type<T>::type*>(diffOutputs),
           nbChannels,
           channelsHeight,
           channelsWidth);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


template void cudaNormalizeL2Forward(const hipDeviceProp_t& deviceProp,
                                   half_float::half alpha,
                                   half_float::half* inputs,
                                   unsigned int nbChannels,
                                   unsigned int channelsHeight,
                                   unsigned int channelsWidth,
                                   unsigned int batchSize,
                                   half_float::half beta,
                                   half_float::half* outputs,
                                   half_float::half* normData,
                                   unsigned int nbOutputs,
                                   unsigned int outputsHeight,
                                   unsigned int outputsWidth);
template void cudaNormalizeL2Forward(const hipDeviceProp_t& deviceProp,
                                   float alpha,
                                   float* inputs,
                                   unsigned int nbChannels,
                                   unsigned int channelsHeight,
                                   unsigned int channelsWidth,
                                   unsigned int batchSize,
                                   float beta,
                                   float* outputs,
                                   float* normData,
                                   unsigned int nbOutputs,
                                   unsigned int outputsHeight,
                                   unsigned int outputsWidth);
template void cudaNormalizeL2Forward(const hipDeviceProp_t& deviceProp,
                                   double alpha,
                                   double* inputs,
                                   unsigned int nbChannels,
                                   unsigned int channelsHeight,
                                   unsigned int channelsWidth,
                                   unsigned int batchSize,
                                   double beta,
                                   double* outputs,
                                   double* normData,
                                   unsigned int nbOutputs,
                                   unsigned int outputsHeight,
                                   unsigned int outputsWidth);

template void cudaNormalizeL2Backward(const hipDeviceProp_t& deviceProp,
                                    half_float::half alpha,
                                    half_float::half* outputs,
                                    half_float::half* normData,
                                    half_float::half* diffInputs,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth,
                                    unsigned int batchSize,
                                    half_float::half beta,
                                    half_float::half* diffOutputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth);
template void cudaNormalizeL2Backward(const hipDeviceProp_t& deviceProp,
                                    float alpha,
                                    float* outputs,
                                    float* normData,
                                    float* diffInputs,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth,
                                    unsigned int batchSize,
                                    float beta,
                                    float* diffOutputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth);
template void cudaNormalizeL2Backward(const hipDeviceProp_t& deviceProp,
                                    double alpha,
                                    double* outputs,
                                    double* normData,
                                    double* diffInputs,
                                    unsigned int nbOutputs,
                                    unsigned int outputsHeight,
                                    unsigned int outputsWidth,
                                    unsigned int batchSize,
                                    double beta,
                                    double* diffOutputs,
                                    unsigned int nbChannels,
                                    unsigned int channelsHeight,
                                    unsigned int channelsWidth);

}
