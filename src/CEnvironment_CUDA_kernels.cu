#include "hip/hip_runtime.h"
/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): Johannes THIELE (johannes.thiele@cea.fr)
                    Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/


#include "CEnvironment_CUDA_kernels.hpp"

//#include "cuPrintf.cu"
#include <stdio.h>




__global__ void cudaNoConversion_kernel(float * data,
                                        float * tickData,
                                        float * tickActivity,
                                        float scaling,
                                        unsigned int inputDimX,
                                        unsigned int inputDimY,
                                        unsigned int inputDimZ)
{
    const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
    const unsigned int batchOffset = blockIdx.x * inputSize;

    for (unsigned int idx = threadIdx.x; idx < inputSize; idx += blockDim.x) {
        float value = data[idx + batchOffset];
        tickData[idx + batchOffset] = scaling*value;
        tickActivity[idx + batchOffset] += scaling*value;
    }
}


__global__ void cudaGenerateInitialSpikes_kernel(float * data,
                                            unsigned long long int * nextEventTime,
                                            int * nextEventType,
                                            unsigned int inputDimX,
                                            unsigned int inputDimY,
                                            unsigned int inputDimZ,
                                            unsigned long long int start,
                                            unsigned long long int stop,
                                            float discardedLateStimuli,
                                            unsigned int stimulusType,
                                            unsigned long long int periodMeanMin,
                                            unsigned long long int periodMeanMax,
                                            float periodRelStdDev,
                                            unsigned long long int periodMin,
                                            float maxFrequency,
                                            hiprandState * state)
{
    const unsigned int inputStride = blockDim.x;
    const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
    const unsigned int batchOffset = blockIdx.x * inputSize;

    // Set local state for performance
    hiprandState local_state = state[threadIdx.x + blockIdx.x * blockDim.x];

    for (unsigned int idx = threadIdx.x; idx < inputSize; idx += inputStride) {

        float value = data[idx + batchOffset];
        int sign = value < 0 ? -1 : 1;

        unsigned long long int eventTime = nextEventTime[idx + batchOffset];
        int eventType = nextEventType[idx + batchOffset];

        /// Include SpikeGenerator::nextEvent in the kernel
        const double delay = 1.0 - fabsf(value);

        const double freq = std::fabs(value) * maxFrequency;

        // TODO: Check if singleBurst is really working properly
        if (delay <= discardedLateStimuli) {
            // SingleBurst
            if (stimulusType == 0) {
                if (eventType == 0) {
                    // High pixel values spike earlier
                    //const double transformDelay = 1.0-fabsf(value)*fabsf(value)*fabsf(value);

                    const unsigned long long int t =
                    (unsigned long long int )(start + delay
                                              * (stop - start));
                    eventTime = t;
                    eventType = sign;
                }
                else {
                    eventTime = 0;
                    eventType = 0;
                }
            }
            else {

                const float freqMeanMax = 1.0 / periodMeanMin;
                const float freqMeanMin = 1.0 / periodMeanMax;
                // value = 0 => most significant => maximal frequency (or minimal
                // period)
                const unsigned long long int  periodMean =
                    (unsigned long long int )(1.0 / (freqMeanMax +
                    (freqMeanMin - freqMeanMax) * delay));

                unsigned long long int t = eventTime;
                unsigned long long int dt = 0;


                // Poissonian
                if (stimulusType == 3){
                    dt = (unsigned long long int)
                            (-logf(hiprand_uniform(&local_state))*periodMean);
                }
                else if (stimulusType == 4) {
                    if (freq >= 1.0/(stop-start)){
                        dt = (unsigned long long int) std::llround(1.0/freq);
                    }
                    else
                        dt = stop + 1;
                }
                else {
                    dt = (unsigned long long int) (hiprand_normal(&local_state) *
                        (periodMean * periodRelStdDev)+periodMean);
                    // JitteredPeriodic
                    if (stimulusType == 2 && (eventType == 0)){

                        dt *= hiprand_uniform(&local_state);
                    }
                }

                if (t > start && dt < periodMin) {
                    dt = periodMin;
                }

                t += dt;

                if (t <= stop) {
                    eventTime = t;
                    eventType = sign;
                }
                else {
                    eventTime = 0;
                    eventType = 0;
                }
            }
            nextEventTime[idx + batchOffset] = eventTime;
            nextEventType[idx + batchOffset] = eventType;
        }

        /// End SpikeGenerator::nextEvent

    }

    // Save current state in global memory between kernel launches
    state[threadIdx.x + blockIdx.x * blockDim.x] = local_state;

}


__global__ void cudaGenerateSpikes_kernel(float * data,
                                            float * tickData,
                                            //float * tickOutputs,
                                            unsigned long long int * nextEventTime,
                                            int * nextEventType,
                                            unsigned int inputDimX,
                                            unsigned int inputDimY,
                                            unsigned int inputDimZ,
                                            unsigned long long int timestamp,
                                            unsigned long long int start,
                                            unsigned long long int stop,
                                            float discardedLateStimuli,
                                            unsigned int stimulusType,
                                            unsigned long long int periodMeanMin,
                                            unsigned long long int periodMeanMax,
                                            float periodRelStdDev,
                                            unsigned long long int periodMin,
                                            float maxFrequency,
                                            hiprandState * state)
{
    const unsigned int inputStride = blockDim.x;
    const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
    const unsigned int batchOffset = blockIdx.x * inputSize;

    // Set local state for performance
    hiprandState local_state = state[threadIdx.x + blockIdx.x * blockDim.x];

    for (unsigned int idx = threadIdx.x; idx < inputSize; idx += inputStride) {

        float value = data[idx + batchOffset];
        int sign = value < 0 ? -1 : 1;

        if (nextEventType[idx + batchOffset] != 0 &&
        nextEventTime[idx + batchOffset] <= timestamp) {
            tickData[idx + batchOffset] = nextEventType[idx + batchOffset];
           
            unsigned long long int eventTime;
            int eventType;

            // This loops creates the next event
            for (unsigned int k = 0; nextEventType[idx + batchOffset] != 0
            && nextEventTime[idx + batchOffset] <= timestamp; ++k) {
                // k>1 if the next event is a spike and still in this time window

                eventTime = nextEventTime[idx + batchOffset];
                eventType = nextEventType[idx + batchOffset];

                /// Include SpikeGenerator::nextEvent in the kernel
                const float delay = 1.0 - fabsf(value);

                const double freq = std::fabs(value) * maxFrequency;

                if (delay <= discardedLateStimuli) {
                    // SingleBurst
                    if (stimulusType == 0) {
                        if (eventType == 0) {
                            // High pixel values spike earlier
                            const unsigned long long int t =
                            (unsigned long long int )(start + delay
                                                      * (stop - start));
                            eventTime = t;
                            eventType = sign;
                        }
                        else {
                            eventTime = 0;
                            eventType = 0;
                        }
                    }
                    else {

                        const float freqMeanMax = 1.0 / periodMeanMin;
                        const float freqMeanMin = 1.0 / periodMeanMax;
                        // value = 0 => most significant => maximal frequency (or minimal
                        // period)
                        const unsigned long long int  periodMean =
                            (unsigned long long int)(1.0 / (freqMeanMax +
                            (freqMeanMin - freqMeanMax) * delay));

                        unsigned long long int t = eventTime;
                        unsigned long long int dt = 0;


                        // Poissonian
                        if (stimulusType == 3){
                            dt = (unsigned long long int)
                                    (-logf(hiprand_uniform(&local_state))*periodMean);
                        }
                        else if (stimulusType == 4) {
                            if (freq >= 1.0/(stop-start)){
                                dt = (unsigned long long int) std::llround(1.0/freq);
                            }
                            else
                                dt = stop + 1;
                        }
                        else {
                            dt = (unsigned long long int) (hiprand_normal(&local_state) *
                                (periodMean * periodRelStdDev)+periodMean);
                            // JitteredPeriodic
                            if (stimulusType == 2 && (eventType == 0)){

                                dt *= hiprand_uniform(&local_state);
                            }

                        }

                        if (t > start && dt < periodMin) {
                            dt = periodMin;
                        }

                        t += dt;

                        if (t <= stop) {
                            eventTime = t;
                            eventType = sign;
                        }

                        else {
                            eventTime = 0;
                            eventType = 0;
                        }
                    }
                    nextEventTime[idx + batchOffset] = eventTime;
                    nextEventType[idx + batchOffset] = eventType;
                }

                /// End SpikeGenerator::nextEvent
            }
        }
        else {
            tickData[idx + batchOffset] = 0;
        }
    }

    // Save current state in global memory between kernel launches
    state[threadIdx.x + blockIdx.x * blockDim.x] = local_state;

}

__global__ void cudaSetupRng_kernel(hiprandState * state, unsigned int seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread gets the same seed but a different sequence number
    hiprand_init(seed, id, 0, &state[id]);
}


void N2D2::cudaNoConversion(float * data,
                            float * tickData,
                            float * tickActivity,
                            float scaling,
                            unsigned int inputsDimX,
                            unsigned int inputsDimY,
                            unsigned int inputsDimZ,
                            unsigned int nbBatches,
                            unsigned int maxNbThreads)
{
    const unsigned int groupSize =
        inputsDimX * inputsDimY * inputsDimZ < maxNbThreads ?
        inputsDimX * inputsDimY * inputsDimZ : maxNbThreads;

    const dim3 blocksPerGrid = {nbBatches, 1, 1};
    const dim3 threadsPerBlocks = {groupSize, 1, 1};

    cudaNoConversion_kernel <<<blocksPerGrid, threadsPerBlocks>>>(data,
                                tickData,
                                tickActivity,
                                scaling,
                                inputsDimX,
                                inputsDimY,
                                inputsDimZ);
}



void N2D2::cudaGenerateInitialSpikes(float * data,
                                unsigned long long int * nextEventTime,
                                int * nextEventType,
                                unsigned int inputDimX,
                                unsigned int inputDimY,
                                unsigned int inputDimZ,
                                unsigned long long int start,
                                unsigned long long int stop,
                                float discardedLateStimuli,
                                unsigned int stimulusType,
                                unsigned long long int periodMeanMin,
                                unsigned long long int periodMeanMax,
                                float periodRelStdDev,
                                unsigned long long int periodMin,
                                float maxFrequency,
                                unsigned int nbBatches,
                                hiprandState * state)
{

    // TODO: Replace 16 by inputSize?
    cudaGenerateInitialSpikes_kernel <<<nbBatches, 16>>>
                                (data,
                                nextEventTime,
                                nextEventType,
                                inputDimX,
                                inputDimY,
                                inputDimZ,
                                start,
                                stop,
                                discardedLateStimuli,
                                stimulusType,
                                periodMeanMin,
                                periodMeanMax,
                                periodRelStdDev,
                                periodMin,
                                maxFrequency,
                                state);
}



void N2D2::cudaGenerateSpikes(float * data,
                                float * tickData,
                                //float * tickOutputs,
                                unsigned long long int * nextEventTime,
                                int * nextEventType,
                                unsigned int inputDimX,
                                unsigned int inputDimY,
                                unsigned int inputDimZ,
                                unsigned long long int timestamp,
                                unsigned long long int start,
                                unsigned long long int stop,
                                float discardedLateStimuli,
                                unsigned int stimulusType,
                                unsigned long long int periodMeanMin,
                                unsigned long long int periodMeanMax,
                                float periodRelStdDev,
                                unsigned long long int periodMin,
                                float maxFrequency,
                                unsigned int nbBatches,
                                hiprandState * state)
{


    cudaGenerateSpikes_kernel <<<nbBatches, 16>>>
                                (data,
                                tickData,
                                //tickOutputs,
                                nextEventTime,
                                nextEventType,
                                inputDimX,
                                inputDimY,
                                inputDimZ,
                                timestamp,
                                start,
                                stop,
                                discardedLateStimuli,
                                stimulusType,
                                periodMeanMin,
                                periodMeanMax,
                                periodRelStdDev,
                                periodMin,
                                maxFrequency,
                                state);
}

void N2D2::cudaSetupRng(hiprandState *state,
                        unsigned int seed,
                        unsigned int nbBatches)
{
    cudaSetupRng_kernel<<<nbBatches, 16>>>(state, seed);
}




