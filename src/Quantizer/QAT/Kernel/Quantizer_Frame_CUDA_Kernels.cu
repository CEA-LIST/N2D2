#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020 CEA LIST. All Rights Reserved.
 *  Contributor(s): Johannes THIELE (johannes.thiele@cea.fr)
 *                  David BRIAND (david.briand@cea.fr)
 *                  Inna KUCHER (inna.kucher@cea.fr)
 *                  Olivier BICHLER (olivier.bichler@cea.fr)
 *                  Vincent TEMPLIER (vincent.templier@cea.fr)
 * 
 * This software is governed by the CeCILL-C license under French law and
 * abiding by the rules of distribution of free software.  You can  use,
 * modify and/ or redistribute the software under the terms of the CeCILL-C
 * license as circulated by CEA, CNRS and INRIA at the following URL
 * "http://www.cecill.info".
 * 
 * As a counterpart to the access to the source code and  rights to copy,
 * modify and redistribute granted by the license, users are provided only
 * with a limited warranty  and the software's author,  the holder of the
 * economic rights,  and the successive licensors  have only  limited
 * liability.
 * 
 * The fact that you are presently reading this means that you have had
 * knowledge of the CeCILL-C license and that you accept its terms.
 * 
 */

#include "Quantizer/QAT/Kernel/Quantizer_Frame_CUDA_Kernels.hpp"
#include <thrust/iterator/constant_iterator.h>
#include "CudaUtils.hpp"
#include <stdlib.h>
#include <math.h>

/* Macros */
#define imin(a,b) (a<b?a:b)


__global__ void cudaH_sum_kernel(__half* x,  
                                 __half* sum, 
                                 unsigned int size)
{
    //256 - threadsPerBlock
    __shared__ __half cache[256];

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    int cacheIndex = threadIdx.x;

    __half temp = __float2half(0.0f);
    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
    temp = __hadd(temp, x[i]);
#else
    temp = __float2half(__half2float(temp) + __half2float(x[i]));
#endif
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0){
        if (cacheIndex < i){
#if __CUDA_ARCH__ >= 530
    cache[cacheIndex] = __hadd(cache[cacheIndex], cache[cacheIndex+i]);
#else
    cache[cacheIndex] = __float2half(__half2float(cache[cacheIndex]) + __half2float(cache[cacheIndex+i]));
#endif
        }
        __syncthreads();
        i /= 2;
    }

    if(cacheIndex == 0){
        sum[blockIdx.x] = __float2half(0.0f);
        sum[blockIdx.x] = cache[0];
    }
}


__global__ void cudaH_variance_kernel(__half* x,
                                      __half* sum, 
                                      __half mean, 
                                      unsigned int size)
{
    //256 - threadsPerBlock
    __shared__ __half cache[256];

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    int cacheIndex = threadIdx.x;

    __half temp = __float2half(0.0f);
    for (unsigned int i = index; i < size; i += stride) {
        //wVariance += (weightsQ(i) - wMean)*(weightsQ(i) - wMean);
#if __CUDA_ARCH__ >= 530
    temp = __hadd(temp, __hmul(__hsub(x[i],mean),__hsub(x[i],mean)));
#else
    temp = __float2half(__half2float(temp) + (__half2float(x[i])-__half2float(mean))*(__half2float(x[i])-__half2float(mean)));
    //temp = __float2half(__half2float(temp) + __half2float(x[i]));
#endif
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0){
        if (cacheIndex < i){
#if __CUDA_ARCH__ >= 530
    cache[cacheIndex] = __hadd(cache[cacheIndex], cache[cacheIndex+i]);
#else
    cache[cacheIndex] = __float2half(__half2float(cache[cacheIndex]) + __half2float(cache[cacheIndex+i]));
#endif
        }
        __syncthreads();
        i /= 2;
    }

    if(cacheIndex == 0){
        sum[blockIdx.x] = __float2half(0.0f);
        sum[blockIdx.x] = cache[0];
    }
}


half_float::half N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_mean(half_float::half* data, 
                                                                half_float::half* partialSum, 
                                                                const unsigned int size)
{
    // return cudaH_accumulate(data, partialSum, size) / half_float::half(size);
    return half_float::half(size);
}


float N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_mean(float* data, 
                                                     const unsigned int size)
{
    return cudaF_accumulate(data, size) / (float)size;
}


double N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_mean(double* data, 
                                                      const unsigned int size)
{
    return cudaD_accumulate(data, size) / (double)size;
}


half_float::half N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_variance(half_float::half* data, 
                                                                    half_float::half* partialSum, 
                                                                    half_float::half mean,
                                                                    const unsigned int size)
{
    int threadsPerBlock = 256;  // Should not be changed
    int blocksPerGrid = imin(32, (size + threadsPerBlock-1) / threadsPerBlock);

    cudaH_variance_kernel<<< (size + 255) / 256, 256>>>(reinterpret_cast<__half*>(data),
                                                        reinterpret_cast<__half*>(partialSum),
                                                        reinterpret_cast<__half&>(mean),
                                                        size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());

    half_float::half* hostPartSumVar = (half_float::half*)malloc(blocksPerGrid*sizeof(half_float::half));
    CHECK_CUDA_STATUS(hipMemcpy(hostPartSumVar,
                                 partialSum,
                                 blocksPerGrid*sizeof(half_float::half),
                                 hipMemcpyDeviceToHost));

    half_float::half sum_var = (half_float::half)0.0f;
    for (int i = 0; i<blocksPerGrid; i++){
        sum_var += hostPartSumVar[i];
    }
    half_float::half variance = sum_var/(half_float::half)(size - 1);

    free(hostPartSumVar);
    return variance;
}

// Calculate Variance var(w)
// https://stackoverflow.com/questions/12380966/standard-deviation-using-cuda
struct thrustF_meanSquare
    : std::unary_function<float, float>
{
    thrustF_meanSquare(float m)
        : mean(m)
    { /* no-op */ }
    const float mean;
    __device__ float operator()(float data) const
    {
        return ::pow(data - mean, 2.0f);
    }
};

float N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_variance(float* data, 
                                                         float mean, 
                                                         const unsigned int size)
{
    thrust::device_ptr<float> dataPtr(data);
    float result = thrust::transform_reduce(dataPtr,
                                            dataPtr+size,
                                            thrustF_meanSquare(mean),
                                            0.0f,
                                            thrust::plus<float>());
    CHECK_CUDA_STATUS(hipPeekAtLastError());

    // To improve the result of the variance, it is recommended 
    // to apply the Bessel’s Correction
    // https://towardsdatascience.com/why-sample-variance-is-divided-by-n-1-89821b83ef6d
    return result / (float)(size - 1);
}


struct thrustD_meanSquare
: std::unary_function<double, double>
{
    thrustD_meanSquare(double m)
    : mean(m)
    { /* no-op */ }
    const double mean;
    __device__ double operator()(double data) const
    {
        return ::pow(data - mean, 2.0);
    }
};

double N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_variance(double* data, 
                                                          double mean, 
                                                          const unsigned int size)
{
    thrust::device_ptr<double> dataPtr(data);
    double result = thrust::transform_reduce(dataPtr,
                                             dataPtr+size,
                                             thrustF_meanSquare(mean),
                                             0.0,
                                             thrust::plus<double>());
    CHECK_CUDA_STATUS(hipPeekAtLastError());

    // To improve the result of the variance, it is recommended 
    // to apply the Bessel’s Correction
    // https://towardsdatascience.com/why-sample-variance-is-divided-by-n-1-89821b83ef6d
    return result / (double)(size - 1);
}


half_float::half N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_accumulate(half_float::half* data, 
                                                                      half_float::half* partialSum, 
                                                                      const unsigned int size)
{
    int threadsPerBlock = 256;  // Should not be changed
    int blocksPerGrid = imin(32, (size + threadsPerBlock-1) / threadsPerBlock);

    cudaH_sum_kernel<<< (size + 255) / 256, 256>>>(reinterpret_cast<__half*>(data),
                                                   reinterpret_cast<__half*>(partialSum),
                                                   size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());

    half_float::half* hostPartSum = (half_float::half*)malloc(blocksPerGrid*sizeof(half_float::half));
    CHECK_CUDA_STATUS(hipMemcpy(hostPartSum,
                                 partialSum,
                                 blocksPerGrid*sizeof(half_float::half),
                                 hipMemcpyDeviceToHost));

    half_float::half sum = (half_float::half)0.0f;
    for (int i = 0; i<blocksPerGrid; ++i){
        sum += hostPartSum[i];
    }   
    free(hostPartSum); 
    return sum;
}


// half_float::half N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_accumulate(half_float::half* data, 
//                                                                       const unsigned int size)
// {
//     int threadsPerBlock = 256;  // Should not be changed
//     int blocksPerGrid = imin(32, (size + threadsPerBlock-1) / threadsPerBlock);

//     __half* partialSum;
//     hipMalloc(&partialSum, blocksPerGrid*sizeof(__half));

//     cudaH_sum_kernel<<< (size + threadsPerBlock-1) / threadsPerBlock, threadsPerBlock>>>(reinterpret_cast<__half*>(data),
//                                                                                          partialSum,
//                                                                                          size);
//     CHECK_CUDA_STATUS(hipPeekAtLastError());

//     half_float::half* hostPartSum = (half_float::half*)malloc(blocksPerGrid*sizeof(half_float::half));
//     CHECK_CUDA_STATUS(hipMemcpy(hostPartSum,
//                                  partialSum,
//                                  blocksPerGrid*sizeof(half_float::half),
//                                  hipMemcpyDeviceToHost));

//     half_float::half sum = (half_float::half)0.0f;
//     for (int i = 0; i<blocksPerGrid; ++i){
//         sum += hostPartSum[i];
//     }   
//     free(hostPartSum); 
//     return sum;
// }


float N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_accumulate(float* data, 
                                                           const unsigned int size)
{
    thrust::device_ptr<float> dataPtr(data);
    return thrust::reduce(dataPtr, dataPtr+size, float(0.0));
}


double N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_accumulate(double* data, 
                                                            const unsigned int size)
{
    thrust::device_ptr<double> dataPtr(data);
    return thrust::reduce(dataPtr, dataPtr+size, double(0.0));
}


__global__ void cudaH_copyData_kernel(__half* x,
                                      __half* y,
                                      unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        __half value = x[i];
        y[i] = value;
#else
        float x_f = __half2float(x[i]);
        float value_f = x_f;
        y[i] = __float2half(value_f);
#endif
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_copyData(half_float::half* input, 
                                                        half_float::half* output, 
                                                        unsigned int inputSize)
{
    cudaH_copyData_kernel<<< (inputSize + 255) / 256, 256>>> (reinterpret_cast<__half*> (input), 
                                                              reinterpret_cast<__half*> (output), 
                                                              inputSize);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


__global__ void cudaF_copyData_kernel(float* x,
                                      float* y,
                                      unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        float value = x[i];
        y[i] = value;
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_copyData(float* input, 
                                                        float* output, 
                                                        unsigned int inputSize)
{
    cudaF_copyData_kernel<<< (inputSize + 255) / 256, 256>>> (input, output, inputSize);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


__global__ void cudaD_copyData_kernel(double* x,
                                      double* y,
                                      unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        double value = x[i];
        y[i] = value;
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_copyData(double* input, 
                                                        double* output, 
                                                        unsigned int inputSize)
{
    cudaD_copyData_kernel<<< (inputSize + 255) / 256, 256>>> (input, output, inputSize);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


struct cudaH_HalfLess : public std::binary_function<__half, __half, bool> 
{
    __device__ bool operator()(const __half& left, const __half& right) const
    {
#if __CUDA_ARCH__ >= 530
        return __hlt(left, right);
#else
        return (__half2float(left) < __half2float(right));
#endif
    }
};

std::pair<half_float::half, half_float::half>
N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_MinMax(half_float::half* data,
                                                 unsigned int size)
{
    thrust::device_ptr<__half> thrustPtr(reinterpret_cast<__half*>(data));
    thrust::pair<thrust::device_vector<__half>::iterator,
                 thrust::device_vector<__half>::iterator> minMaxPair
        = thrust::minmax_element(thrustPtr, thrustPtr + size, cudaH_HalfLess());

    const __half minVal = *(minMaxPair.first);
    const __half maxVal = *(minMaxPair.second);

    return std::make_pair(reinterpret_cast<const half_float::half&>(minVal),
                          reinterpret_cast<const half_float::half&>(maxVal));
}

std::pair<float, float> 
N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_MinMax(float* data, 
                                                 unsigned int size)
{
    thrust::device_ptr<float> thrustPtr(data);
    thrust::pair<thrust::device_ptr<float>, thrust::device_ptr<float>> 
            minMaxPair = thrust::minmax_element(thrustPtr, thrustPtr+size);

    return std::make_pair(*(minMaxPair.first), *(minMaxPair.second));
}

std::pair<double, double> 
N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_MinMax(double* data, 
                                                 unsigned int size)
{
    thrust::device_ptr<double> thrustPtr(data);
    thrust::pair<thrust::device_ptr<double>, thrust::device_ptr<double>> 
            minMaxPair = thrust::minmax_element(thrustPtr, thrustPtr+size);
            
    return std::make_pair(*(minMaxPair.first), *(minMaxPair.second));
}


__global__ void cudaDivH_kernel(__half* data,
                                size_t size,
                                __half value)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    float div_value = 1/(float)value;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530 && defined(CUDART_VERSION) && CUDART_VERSION >= 8000
        data[i] = __hmul(data[i], __float2half(div_value));
#else
        data[i] = __float2half(__half2float(data[i]) * div_value);
#endif
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_div(half_float::half* data,
                                                   unsigned int size, 
                                                   half_float::half value)
{
    cudaDivH_kernel<<<(size + 255) / 256, 256>>>(reinterpret_cast<__half*>(data), 
                                                 size, 
                                                 reinterpret_cast<__half&>(value));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


void N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_div(float* data, 
                                                   unsigned int size, 
                                                   float value)
{
    thrust::device_ptr<float> thrustPtr(data);
    thrust::transform(thrustPtr, 
                      thrustPtr + size,
                      thrust::make_constant_iterator((float)value), 
                      thrustPtr,
                      thrust::divides<float>());
}


void N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_div(double* data, 
                                                   unsigned int size, 
                                                   double value)
{
    thrust::device_ptr<double> thrustPtr(data);
    thrust::transform(thrustPtr, 
                      thrustPtr + size,
                      thrust::make_constant_iterator((double)value), 
                      thrustPtr,
                      thrust::divides<double>());
}


__global__ void cudaH_tanh_kernel(__half* x, 
                                  __half* y, 
                                  unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
        float value_f = __half2float(x[i]);
        y[i] = __float2half(tanh(value_f));
#elif defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 530) 
        float value_f = __half2float(x[i]);
        y[i] = __float2half(tanh(value_f));
#endif
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaH_tanh(half_float::half* input,
                                                    half_float::half* output,
                                                    unsigned int size)
{
    cudaH_tanh_kernel<<< (size + 255) / 256, 256>>>(reinterpret_cast<__half*>(input), 
                                                    reinterpret_cast<__half*>(output), 
                                                    size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


__global__ void cudaF_tanh_kernel(float* x,
                                  float* y,
                                  unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = tanh(x[i]);
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaF_tanh(float* input,
                                                    float* output,
                                                    unsigned int size)
{
    cudaF_tanh_kernel<<< (size + 255) / 256, 256>>>(input, output, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


__global__ void cudaD_tanh_kernel(double* x,
                                  double* y,
                                  unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = tanh(x[i]);
    }
}

void N2D2::Quantizer_Frame_CUDA_Kernels::cudaD_tanh(double* input,
                                                    double* output,
                                                    unsigned int size)
{
    cudaD_tanh_kernel<<< (size + 255) / 256, 256>>>(input, output, size);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}