#include "hip/hip_runtime.h"
/*
    (C) Copyright 2016 CEA LIST. All Rights Reserved.
    Contributor(s): Olivier BICHLER (olivier.bichler@cea.fr)

    This software is governed by the CeCILL-C license under French law and
    abiding by the rules of distribution of free software.  You can  use,
    modify and/ or redistribute the software under the terms of the CeCILL-C
    license as circulated by CEA, CNRS and INRIA at the following URL
    "http://www.cecill.info".

    As a counterpart to the access to the source code and  rights to copy,
    modify and redistribute granted by the license, users are provided only
    with a limited warranty  and the software's author,  the holder of the
    economic rights,  and the successive licensors  have only  limited
    liability.

    The fact that you are presently reading this means that you have had
    knowledge of the CeCILL-C license and that you accept its terms.
*/

#include "Solver/SGDSolver_CUDA_Kernels.hpp"
#include "CudaUtils.hpp"

__global__ void
cudaHclamp_kernel(__half* x, unsigned int size, __half minVal, __half maxVal)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        x[i] = (__hlt(x[i], minVal)) ? minVal :
               (__hgt(x[i], maxVal)) ? maxVal :
                                       x[i];
#else
        x[i] = (__half2float(x[i]) < __half2float(minVal)) ? minVal :
               (__half2float(x[i]) > __half2float(maxVal)) ? maxVal :
                                                             x[i];
#endif
    }
}

__global__ void cudaHquantize_kernel(__half* x,
                                     __half* y,
                                     unsigned int size,
                                     __half minVal,
                                     __half maxVal,
                                     unsigned int quantizationLevels,
                                     bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const float scaling = (__half2float(maxVal) - __half2float(minVal))
            / (float)(quantizationLevels - 1);

        for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
            const __half clamped = (__hlt(x[i], minVal)) ? minVal :
                                   (__hgt(x[i], maxVal)) ? maxVal :
                                                           x[i];
#else
            const __half clamped
                = (__half2float(x[i]) < __half2float(minVal)) ? minVal :
                  (__half2float(x[i]) > __half2float(maxVal)) ? maxVal :
                                                                x[i];
#endif

            if (truncate) {
                y[i] = __float2half(
                    (int)((__half2float(clamped) - __half2float(minVal))
                               / scaling) * scaling + __half2float(minVal));
            }
            else {
                y[i] = __float2half(
                    (int)round((__half2float(clamped) - __half2float(minVal))
                               / scaling) * scaling + __half2float(minVal));
            }
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride)
            y[i] = __float2half((__half2float(x[i]) >= 0.0f) ? 1.0f : -1.0f);
    }
}

__global__ void
cudaSclamp_kernel(float* x, unsigned int size, float minVal, float maxVal)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        x[i] = (x[i] < minVal) ? minVal :
               (x[i] > maxVal) ? maxVal :
                                 x[i];
    }
}

__global__ void cudaSquantize_kernel(float* x,
                                     float* y,
                                     unsigned int size,
                                     float minVal,
                                     float maxVal,
                                     unsigned int quantizationLevels,
                                     bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const float scaling = (maxVal - minVal)
            / (float)(quantizationLevels - 1);

        for (unsigned int i = index; i < size; i += stride) {
            const float clamped = (x[i] < minVal) ? minVal :
                                  (x[i] > maxVal) ? maxVal :
                                                    x[i];

            if (truncate)
                y[i] = (int)((clamped - minVal) / scaling) * scaling + minVal;
            else {
                y[i] = (int)round((clamped - minVal) / scaling)
                        * scaling + minVal;
            }
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride)
            y[i] = ((x[i] >= 0.0f) ? 1.0f : -1.0f);
    }
}

__global__ void
cudaDclamp_kernel(double* x, unsigned int size, double minVal, double maxVal)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        x[i] = (x[i] < minVal) ? minVal :
               (x[i] > maxVal) ? maxVal :
                                 x[i];
    }
}

__global__ void cudaDquantize_kernel(double* x,
                                     double* y,
                                     unsigned int size,
                                     double minVal,
                                     double maxVal,
                                     unsigned int quantizationLevels,
                                     bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const double scaling = (maxVal - minVal)
            / (double)(quantizationLevels - 1);

        for (unsigned int i = index; i < size; i += stride) {
            const double clamped = (x[i] < minVal) ? minVal :
                                  (x[i] > maxVal) ? maxVal :
                                                    x[i];

            if (truncate)
                y[i] = (int)((clamped - minVal) / scaling) * scaling + minVal;
            else {
                y[i] = (int)round((clamped - minVal) / scaling)
                        * scaling + minVal;
            }
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride)
            y[i] = ((x[i] >= 0.0) ? 1.0 : -1.0);
    }
}

__global__ void cudaHscal_kernel(unsigned int size,
                                 __half alpha,
                                 __half *x)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        x[i] = __hmul(alpha, x[i]);
#else
        x[i] = __float2half(__half2float(alpha) * __half2float(x[i]));
#endif
    }
}

__global__ void cudaHaxpy_kernel(unsigned int size,
                                 __half alpha,
                                 const __half *x,
                                 __half *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd(__hmul(alpha, x[i]), y[i]);
#else
        y[i] = __float2half(__half2float(alpha) * __half2float(x[i])
                            + __half2float(y[i]));
#endif
    }
}

__global__ void cudaHpow_kernel(unsigned int size,
                                 __half power,
                                 const __half *x,
                                 __half *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = __float2half(powf(__half2float(x[i]), __half2float(power)));
    }
}

__global__ void cudaSpow_kernel(unsigned int size,
                                 float power,
                                 const float *x,
                                 float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = powf(x[i], power);
    }
}

__global__ void cudaDpow_kernel(unsigned int size,
                                 double power,
                                 const double *x,
                                 double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = powf(x[i], power);
    }
}

__global__ void cudaHadd_kernel(unsigned int size,
                                 __half value,
                                 const __half *x,
                                 __half *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hadd(x[i], value);
#else
        y[i] = __float2half(__half2float(x[i]) + __half2float(value));
#endif
    }
}

__global__ void cudaSadd_kernel(unsigned int size,
                                 float value,
                                 const float *x,
                                 float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = x[i] + value;
    }
}

__global__ void cudaDadd_kernel(unsigned int size,
                                 double value,
                                 const double *x,
                                 double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = x[i] + value;
    }
}

__global__ void cudaHmult_kernel(unsigned int size,
                                 const __half *x1,
                                 const __half *x2,
                                 __half *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
#if __CUDA_ARCH__ >= 530
        y[i] = __hmul(x1[i], x2[i]);
#else
        y[i] = __float2half(__half2float(x1[i]) + __half2float(x2[i]));
#endif
    }
}

__global__ void cudaSmult_kernel(unsigned int size,
                                 const float *x1,
                                 const float *x2,
                                 float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = x1[i] * x2[i];
    }
}

__global__ void cudaDmult_kernel(unsigned int size,
                                 const double *x1,
                                 const double *x2,
                                 double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = x1[i] * x2[i];
    }
}

__global__ void cudaHinv_kernel(unsigned int size,
                                 const __half *x,
                                 __half *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = __float2half(1.0f / __half2float(x[i]));
    }
}

__global__ void cudaSinv_kernel(unsigned int size,
                                 const float *x,
                                 float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = 1.0f / x[i];
    }
}

__global__ void cudaDinv_kernel(unsigned int size,
                                 const double *x,
                                 double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        y[i] = 1.0 / x[i];
    }
}

void N2D2::cudaHclamp(half_float::half* x, unsigned int size,
                      half_float::half minVal, half_float::half maxVal)
{
    cudaHclamp_kernel<<<(size + 255) / 256, 256>>>(reinterpret_cast<__half*>(x),
                                            size,
                                            reinterpret_cast<__half&>(minVal),
                                            reinterpret_cast<__half&>(maxVal));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

struct HalfLess : public std::binary_function<__half, __half, bool> {
    __device__ bool operator()(const __half& left, const __half& right) const
    {
#if __CUDA_ARCH__ >= 530
        return __hlt(left, right);
#else
        return (__half2float(left) < __half2float(right));
#endif
    }
};

std::pair<half_float::half, half_float::half>
N2D2::cudaHminMax(half_float::half* x,
                  unsigned int size)
{
    // Compute global min & max value on the full tensor
    thrust::device_ptr<__half> thrustPtr(reinterpret_cast<__half*>(x));
    thrust::pair<thrust::device_vector<__half>::iterator,
                 thrust::device_vector<__half>::iterator> minMaxPair
        = thrust::minmax_element(thrustPtr, thrustPtr + size, HalfLess());

    const __half minVal = *(minMaxPair.first);
    const __half maxVal = *(minMaxPair.second);

    return std::make_pair(reinterpret_cast<const half_float::half&>(minVal),
                          reinterpret_cast<const half_float::half&>(maxVal));
}

void N2D2::cudaHquantize(half_float::half* x,
                         half_float::half* y,
                         unsigned int size,
                         half_float::half minVal,
                         half_float::half maxVal,
                         unsigned int quantizationLevels,
                         bool truncate)
{
    cudaHquantize_kernel<<<(size + 255) / 256, 256>>>
        (reinterpret_cast<__half*>(x),
         reinterpret_cast<__half*>(y),
         size,
         reinterpret_cast<__half&>(minVal),
         reinterpret_cast<__half&>(maxVal),
         quantizationLevels,
         truncate);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSclamp(float* x, unsigned int size, float minVal, float maxVal)
{
    cudaSclamp_kernel<<<(size + 255) / 256, 256>>>(x, size, minVal, maxVal);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

std::pair<float, float>
N2D2::cudaSminMax(float* x,
                  unsigned int size)
{
    // Compute global min & max value on the full tensor
    thrust::device_ptr<float> thrustPtr(x);
    thrust::pair<thrust::device_vector<float>::iterator,
                 thrust::device_vector<float>::iterator> minMaxPair
        = thrust::minmax_element(thrustPtr, thrustPtr + size);

    return std::make_pair(*(minMaxPair.first), *(minMaxPair.second));
}

void N2D2::cudaSquantize(float* x,
                         float* y,
                         unsigned int size,
                         float minVal,
                         float maxVal,
                         unsigned int quantizationLevels,
                         bool truncate)
{
    cudaSquantize_kernel<<<(size + 255) / 256, 256>>>
        (x, y, size, minVal, maxVal, quantizationLevels, truncate);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void
N2D2::cudaDclamp(double* x, unsigned int size, double minVal, double maxVal)
{
    cudaDclamp_kernel<<<(size + 255) / 256, 256>>>
        (x, size, minVal, maxVal);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

std::pair<double, double>
N2D2::cudaDminMax(double* x,
                  unsigned int size)
{
    // Compute global min & max value on the full tensor
    thrust::device_ptr<double> thrustPtr(x);
    thrust::pair<thrust::device_vector<double>::iterator,
                 thrust::device_vector<double>::iterator> minMaxPair
        = thrust::minmax_element(thrustPtr, thrustPtr + size);

    return std::make_pair(*(minMaxPair.first), *(minMaxPair.second));
}

void N2D2::cudaDquantize(double* x,
                         double* y,
                         unsigned int size,
                         double minVal,
                         double maxVal,
                         unsigned int quantizationLevels,
                         bool truncate)
{
    cudaDquantize_kernel<<<(size + 255) / 256, 256>>>
        (x, y, size, minVal, maxVal, quantizationLevels, truncate);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHscal(unsigned int size,
                     half_float::half alpha,
                     half_float::half *x)
{
    cudaHscal_kernel<<<(size + 255) / 256, 256>>>
        (size,
        reinterpret_cast<__half&>(alpha),
        reinterpret_cast<__half*>(x));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHaxpy(unsigned int size,
                     half_float::half alpha,
                     const half_float::half *x,
                     half_float::half *y)
{
    cudaHaxpy_kernel<<<(size + 255) / 256, 256>>>
        (size,
         reinterpret_cast<__half&>(alpha),
         reinterpret_cast<const __half*>(x),
         reinterpret_cast<__half*>(y));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHpow(unsigned int size,
                      half_float::half power,
                      const half_float::half *x,
                      half_float::half *y)
{
    cudaHpow_kernel<<<(size + 255) / 256, 256>>>
        (size,
        reinterpret_cast<__half&>(power),
        reinterpret_cast<const __half*>(x),
        reinterpret_cast<__half*>(y));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSpow(unsigned int size,
                      float power,
                      const float *x,
                      float *y)
{
    cudaSpow_kernel<<<(size + 255) / 256, 256>>>(size, power, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDpow(unsigned int size,
                      double power,
                      const double *x,
                      double *y)
{
    cudaDpow_kernel<<<(size + 255) / 256, 256>>>(size, power, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHadd(unsigned int size,
                    half_float::half value,
                    const half_float::half *x,
                    half_float::half *y)
{
    cudaHadd_kernel<<<(size + 255) / 256, 256>>>
        (size,
        reinterpret_cast<__half&>(value),
        reinterpret_cast<const __half*>(x),
        reinterpret_cast<__half*>(y));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSadd(unsigned int size,
                      float value,
                      const float *x,
                      float *y)
{
    cudaSadd_kernel<<<(size + 255) / 256, 256>>>(size, value, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDadd(unsigned int size,
                      double value,
                      const double *x,
                      double *y)
{
    cudaDadd_kernel<<<(size + 255) / 256, 256>>>(size, value, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}


void N2D2::cudaHmult(unsigned int size,
                      const half_float::half *x1,
                      const half_float::half *x2,
                      half_float::half *y)
{
    cudaHmult_kernel<<<(size + 255) / 256, 256>>>
        (size,
        reinterpret_cast<const __half*>(x1),
        reinterpret_cast<const __half*>(x2),
        reinterpret_cast<__half*>(y));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSmult(unsigned int size,
                      const float *x1,
                      const float *x2,
                      float *y)
{
    cudaSmult_kernel<<<(size + 255) / 256, 256>>>(size, x1, x2, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDmult(unsigned int size,
                      const double *x1,
                      const double *x2,
                      double *y)
{
    cudaDmult_kernel<<<(size + 255) / 256, 256>>>(size, x1, x2, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaHinv(unsigned int size,
                    const half_float::half *x,
                    half_float::half *y)
{
    cudaHinv_kernel<<<(size + 255) / 256, 256>>>
        (size,
        reinterpret_cast<const __half*>(x),
        reinterpret_cast<__half*>(y));
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaSinv(unsigned int size,
                      const float *x,
                      float *y)
{
    cudaSinv_kernel<<<(size + 255) / 256, 256>>>(size, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}

void N2D2::cudaDinv(unsigned int size,
                      const double *x,
                      double *y)
{
    cudaDinv_kernel<<<(size + 255) / 256, 256>>>(size, x, y);
    CHECK_CUDA_STATUS(hipPeekAtLastError());
}
